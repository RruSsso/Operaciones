#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>

// Addition function on GPU
__global__ void add_gpu(int* a, int* b, int* result)
{
    *result = *a + *b;
}

// Addition function on CPU
void add_cpu(int* a, int* b, int* result)
{
    *result = *a + *b;
}

// Subtraction function on GPU
__global__ void subtract_gpu(int* a, int* b, int* result)
{
    *result = *b - *a;
}

// Subtraction function on CPU
void subtract_cpu(int* a, int* b, int* result)
{
    *result = *b - *a;
}

// Multiplication function on GPU
__global__ void multiply_gpu(int* a, int* b, int* result)
{
    *result = *a * *b;
}

// Multiplication function on CPU
void multiply_cpu(int* a, int* b, int* result)
{
    *result = *a * *b;
}

// Division function on GPU
__global__ void divide_gpu(int* a, int* b, int* result)
{
    if (*a != 0)
        *result = *b / *a;
    else
        *result = 0;  // Avoid division by zero
}

// Division function on CPU
void divide_cpu(int* a, int* b, int* result)
{
    if (*a != 0)
        *result = *b / *a;
    else
        *result = 0;  // Avoid division by zero
}

int main()
{
    int a = 2;
    int b = 5;
    int result;

    // CPU Operations
    std::cout << "CPU Operations:" << std::endl;

    add_cpu(&a, &b, &result);
    std::cout << "Addition (CPU): " << result << std::endl;

    subtract_cpu(&a, &b, &result);
    std::cout << "Subtraction (CPU): " << result << std::endl;

    multiply_cpu(&a, &b, &result);
    std::cout << "Multiplication (CPU): " << result << std::endl;

    divide_cpu(&a, &b, &result);
    std::cout << "Division (CPU): " << result << std::endl;

    // GPU Memory Allocation
    int* a_gpu, * b_gpu, * result_gpu;
    int size = sizeof(int);

    hipMalloc((void**)&a_gpu, size);
    hipMalloc((void**)&b_gpu, size);
    hipMalloc((void**)&result_gpu, size);

    hipMemcpy(a_gpu, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, &b, size, hipMemcpyHostToDevice);

    // GPU Operations
    std::cout << "\nGPU Operations:" << std::endl;

    add_gpu << <1, 1 >> > (a_gpu, b_gpu, result_gpu);
    hipMemcpy(&result, result_gpu, size, hipMemcpyDeviceToHost);
    std::cout << "Addition (GPU): " << result << std::endl;

    subtract_gpu << <1, 1 >> > (a_gpu, b_gpu, result_gpu);
    hipMemcpy(&result, result_gpu, size, hipMemcpyDeviceToHost);
    std::cout << "Subtraction (GPU): " << result << std::endl;

    multiply_gpu << <1, 1 >> > (a_gpu, b_gpu, result_gpu);
    hipMemcpy(&result, result_gpu, size, hipMemcpyDeviceToHost);
    std::cout << "Multiplication (GPU): " << result << std::endl;

    divide_gpu << <1, 1 >> > (a_gpu, b_gpu, result_gpu);
    hipMemcpy(&result, result_gpu, size, hipMemcpyDeviceToHost);
    std::cout << "Division (GPU): " << result << std::endl;

    // Free GPU memory
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(result_gpu);

    return 0;
}
